
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_20 --sysroot=%S/Inputs/CUDA 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=OK
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_20 --sysroot=%S/Inputs/CUDA_80 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=OK
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --sysroot=%S/Inputs/CUDA_80 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=OK

// The installation at Inputs/CUDA is CUDA 7.0, which doesn't support sm_60.
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --sysroot=%S/Inputs/CUDA 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=ERR_SM60

// This should only complain about sm_60, not sm_35.
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_35 \
// RUN:    --sysroot=%S/Inputs/CUDA 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=ERR_SM60 --check-prefix=OK_SM35

// We should get two errors here, one for sm_60 and one for sm_61.
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --cuda-gpu-arch=sm_61 \
// RUN:    --sysroot=%S/Inputs/CUDA 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=ERR_SM60 --check-prefix=ERR_SM61

// We should still get an error if we pass -nocudainc, because this compilation
// would invoke ptxas, and we do a version check on that, too.
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 -nocudainc --sysroot=%S/Inputs/CUDA 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=ERR_SM60

// If with -nocudainc and -E, we don't touch the CUDA install, so we
// shouldn't get an error.
// RUN: %clang --target=x86_64-linux -v -### -E --cuda-device-only --cuda-gpu-arch=sm_60 -nocudainc \
// RUN:    --sysroot=%S/Inputs/CUDA 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=OK

// --no-cuda-version-check should suppress all of these errors.
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --sysroot=%S/Inputs/CUDA 2>&1 \
// RUN:    --no-cuda-version-check %s | \
// RUN:    FileCheck %s --check-prefix=OK

// We need to make sure the version check is done only for the device toolchain,
// therefore we should not get an error in host-only mode. We use the -S here
// to avoid the error being produced in case by the assembler tool, which does
// the same check.
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --cuda-host-only --sysroot=%S/Inputs/CUDA -S 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=OK
// RUN: %clang --target=x86_64-linux -v -### --cuda-gpu-arch=sm_60 --cuda-device-only --sysroot=%S/Inputs/CUDA -S 2>&1 %s | \
// RUN:    FileCheck %s --check-prefix=ERR_SM60

// OK-NOT: error: GPU arch

// OK_SM35-NOT: error: GPU arch sm_35

// We should only get one error per architecture.
// ERR_SM60: error: GPU arch sm_60 {{.*}}
// ERR_SM60-NOT: error: GPU arch sm_60

// ERR_SM61: error: GPU arch sm_61 {{.*}}
// ERR_SM61-NOT: error: GPU arch sm_61
